#include "hip/hip_runtime.h"
﻿//点乘运算
//（a,b,c）*(d,e,f)=a*d+b*e+c*f; 
//warp为32，因此将blocksPerGrid一般设置为32
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#define imin(a,b) (a<b?a:b)
const int N = 2 * 4;
const int threadsPerBlock = 256;
const int blockPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

__global__ void dot(float *a, float *b, float *c)
{
	__shared__ float cache[threadsPerBlock];
	//对于GPU上启动的每个线程块，CUDA C编译器都将创建该共享变量的一个副本。线程块中的每个线程都共享这块内存

	int tid = threadIdx.x + blockDim.x*blockIdx.x;//总索引
	int cacheIndex = threadIdx.x;
	float temp = 0;

	while (tid < N)
	{
		temp += a[tid] + b[tid];
		tid += blockDim.x*gridDim.x;
	}

	cache[cacheIndex] = temp;

	__syncthreads();//保证线程块中的线程都执行完__synthreads()之前的语句

	int i = blockDim.x / 2;
	while (i != 0)
	{
		if (cacheIndex < i) cache[cacheIndex] += cache[cacheIndex + i];
		__syncthreads();
		i /= 2;
	}
	if (cacheIndex == 0)
		c[blockIdx.x] = cache[0];//将每个block内的线程之和保存到c中
}

int main()
{
	float *a, *b, sum = 0, *partial_c;
	float *deva, *devb, *devpartial_c;
	a = new float[N];
	b = new float[N];
	partial_c = new float[blockPerGrid];
	//在GPU上分配内存
	hipMalloc((void **)&deva, N * sizeof(float));
	hipMalloc((void **)&devb, N * sizeof(float));
	hipMalloc((void **)&devpartial_c, blockPerGrid * sizeof(float));

	//在CPU上为数组赋值
	for (int i = 0; i < N; i++)
	{
		a[i] = i;
		b[i] = 2 * i;
	}
	//将数组a和b传到GPU
	hipMemcpy(deva, a, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devb, b, N * sizeof(float), hipMemcpyHostToDevice);

	dot << <blockPerGrid, threadsPerBlock >> >(deva, devb, devpartial_c);

	//将数组c从GPU传到CPU
	hipMemcpy(partial_c, devpartial_c, blockPerGrid * sizeof(float), hipMemcpyDeviceToHost);

	//在CPU上完成最终求和运算
	for (int i = 0; i < blockPerGrid; i++)
		sum += partial_c[i];

	printf("value %g\n", sum);

	hipFree(deva);
	hipFree(devb);
	hipFree(devpartial_c);

	return 0;
}